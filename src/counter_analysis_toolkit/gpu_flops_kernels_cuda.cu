#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <papi.h>
#include "gpu_flops_kernels.h"
#include "gpu_flops.h"

#if defined(GPU_NVIDIA)

#define NUM_SIZES 3
#define NUM_TYPES 6
#define MAX_SIZE  64

static void gpu_matrix_flop(int EventSet, FILE *ofp_papi);

/* Wrapper functions of different precisions. */
#if defined(CAT_GPU_PREC_DP)
    extern "C" void gpu_matrix_flop_dp(int EventSet, FILE *ofp_papi) {
        gpu_matrix_flop(EventSet, ofp_papi);
    }
#elif defined(CAT_GPU_PREC_SP)
    extern "C" void gpu_matrix_flop_sp(int EventSet, FILE *ofp_papi) {
        gpu_matrix_flop(EventSet, ofp_papi);
    }
#elif defined(CAT_GPU_PREC_HP)
    extern "C" void gpu_matrix_flop_hp(int EventSet, FILE *ofp_papi) {
        gpu_matrix_flop(EventSet, ofp_papi);
    }
#endif

static void gpu_matrix_flop(int EventSet, FILE *ofp_papi) {

    int i, j, retval;
    CAT_GPU_PREC junk = 0.0;
    long long values = 0;
    size_t maxSize;
    size_t sizes[NUM_SIZES];
    int sizeIdx, typeIdx;
    int types[NUM_TYPES];
    hipError_t status;

    CAT_GPU_PREC  *hostA,  *hostB,  *hostC;
    CAT_GPU_PREC   *devA,   *devB,   *devC;

    /* Create a list of types. */
    types[0] = ADD;
    types[1] = MUL;
    types[2] = DIV;
    types[3] = SQRT;
    types[4] = SUB;
    types[5] = FMA;

    /* Create a list of sizes. */
    sizes[0] = 16;
    sizes[1] = 32;
    sizes[2] = 64;

    maxSize = MAX_SIZE*MAX_SIZE*sizeof(CAT_GPU_PREC);

    /* Allocate host arrays. */
    status = hipHostMalloc((void**)&hostA, maxSize);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not allocate host matrix A.\n");
        return;
    }
    status = hipHostMalloc((void**)&hostB, maxSize);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not allocate host matrix B.\n");
        return;
    }
    status = hipHostMalloc((void**)&hostC, maxSize);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not allocate host matrix C.\n");
        return;
    }

    /* Allocate device arrays. */
    status = hipMalloc((void**)&devA, maxSize);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not allocate device matrix A.\n");
        return;
    }
    status = hipMalloc((void**)&devB, maxSize);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not allocate device matrix B.\n");
        return;
    }
    status = hipMalloc((void**)&devC, maxSize);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not allocate device matrix C.\n");
        return;
    }

    /* Randomly initialize arrays. */
    srandom(1);
    for( i = 0; i < MAX_SIZE; i++ ) {
        for( j = 0; j < MAX_SIZE; j++ ) {
            hostA[i*MAX_SIZE + j] = ((CAT_GPU_PREC)random())/((CAT_GPU_PREC)RAND_MAX) + (CAT_GPU_PREC)1.1;
            hostB[i*MAX_SIZE + j] = ((CAT_GPU_PREC)random())/((CAT_GPU_PREC)RAND_MAX) + (CAT_GPU_PREC)1.1;
            hostC[i*MAX_SIZE + j] = 0.0;
        }
    }

    /* Copy host data to device. */
    status = hipMemcpy(devA, hostA, maxSize, hipMemcpyHostToDevice);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not copy matrix A to device.\n");
        return;
    }
    status = hipMemcpy(devB, hostB, maxSize, hipMemcpyHostToDevice);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not copy matrix B to device.\n");
        return;
    }
    status = hipMemcpy(devC, hostC, maxSize, hipMemcpyHostToDevice);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not copy matrix C to device.\n");
        return;
    }

    /* Loop through the kernels. */
    for(typeIdx = 0; typeIdx < NUM_TYPES; ++typeIdx) {

        for(sizeIdx = 0; sizeIdx < NUM_SIZES; ++sizeIdx) {

            size_t N = sizes[sizeIdx];
            int type = types[typeIdx];

            /* Device configuration. */
            dim3 threads_per_block( 1, 1, 1 );
            dim3 blocks_in_grid( ceil( float(N) / threads_per_block.x ), ceil( float(N) / threads_per_block.y ), 1 );

            /* Start PAPI counters. */
            if( (retval = PAPI_start( EventSet )) != PAPI_OK ) {
                fprintf(stderr, "GPU FLOPs Benchmark: PAPI_start() returned error code %d\n", retval);
                return;
            }

            /* Various floating-point operation kernels. */
            switch(type) {
              case ADD:
                matrix_add<<<blocks_in_grid, threads_per_block>>>(devA, devB, devC, N);
                break;
              case MUL:
                matrix_mul<<<blocks_in_grid, threads_per_block>>>(devA, devB, devC, N);
                break;
              case DIV:
                matrix_div<<<blocks_in_grid, threads_per_block>>>(devA, devB, devC, N);
                break;
              case SQRT:
                matrix_sqrt<<<blocks_in_grid, threads_per_block>>>(devA, devB, devC, N);
                break;
              case SUB:
                matrix_sub<<<blocks_in_grid, threads_per_block>>>(devA, devB, devC, N);
                break;
              case FMA:
                matrix_fma<<<blocks_in_grid, threads_per_block>>>(devA, devB, devC, N);
                break;
              default:
                break;
            }

            /* Error checking -- consider calling hipGetLastError() after PAPI_stop(). */
            /*status = hipGetLastError();
            if( hipSuccess != status ) {
                fprintf(stderr, "Error 1.\n");
                return;
            }*/
            status = hipDeviceSynchronize();
            if( hipSuccess != status ) {
                fprintf(stderr, "Error 2.\n");
                return;
            }

            /* Stop PAPI counters. */
            if( (retval = PAPI_stop(EventSet, &values)) != PAPI_OK ) {
                fprintf(stderr, "GPU FLOPs Benchmark: PAPI_stop() returned error code %d\n", retval);
                return;
            }
            fprintf(ofp_papi, "%lld\n", values);

        }
    }

    /* Copy device data to host. */
    status = hipMemcpy(hostC, devC, maxSize, hipMemcpyDeviceToHost);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not copy matrix C from device.\n");
        return;
    }

    /* Use the result from the kernels to prevent compiler optimizing it away. */
    junk = ((CAT_GPU_PREC)1.23+hostC[MAX_SIZE*MAX_SIZE/2])/((CAT_GPU_PREC)1.45+hostC[4*MAX_SIZE*MAX_SIZE/5]*hostC[MAX_SIZE*MAX_SIZE-1]);
    if( junk > (CAT_GPU_PREC)1.23 && junk < (CAT_GPU_PREC)1.2345 )
        fprintf(stdout, "Benchmark artifact (%f) -- ignore.\n", (float)junk);

    /* Free device memory. */
    status = hipFree(devA);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not free device matrix A.\n");
        return;
    }
    status = hipFree(devB);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not free device matrix B.\n");
        return;
    }
    status = hipFree(devC);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not free device matrix C.\n");
        return;
    }

    /* Free host memory. */
    status = hipHostFree(hostA);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not free host matrix A.\n");
        return;
    }
    status = hipHostFree(hostB);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not free host matrix B.\n");
        return;
    }
    status = hipHostFree(hostC);
    if( hipSuccess != status ) {
        fprintf(stderr, "Could not free host matrix C.\n");
        return;
    }

    return;
}

__global__ void matrix_add(CAT_GPU_PREC *A, CAT_GPU_PREC *B, CAT_GPU_PREC *C, int N) {

    int colIdx = blockDim.x*blockIdx.x + threadIdx.x;
    int rowIdx = blockDim.y*blockIdx.y + threadIdx.y;

    if( rowIdx < N && colIdx < N ) { /* This check shouldn't be necessary for now. */
        C[rowIdx*N + colIdx] = ADD_INTRIN(A[rowIdx*N + colIdx], B[rowIdx*N + colIdx]);
    }
}

__global__ void matrix_sub(CAT_GPU_PREC *A, CAT_GPU_PREC *B, CAT_GPU_PREC *C, int N) {

    int colIdx = blockDim.x*blockIdx.x + threadIdx.x;
    int rowIdx = blockDim.y*blockIdx.y + threadIdx.y;

    if( rowIdx < N && colIdx < N ) {
        C[rowIdx*N + colIdx] = SUB_INTRIN(A[rowIdx*N + colIdx], B[rowIdx*N + colIdx]);
    }
}

__global__ void matrix_mul(CAT_GPU_PREC *A, CAT_GPU_PREC *B, CAT_GPU_PREC *C, int N) {

    int colIdx = blockDim.x*blockIdx.x + threadIdx.x;
    int rowIdx = blockDim.y*blockIdx.y + threadIdx.y;

    if( rowIdx < N && colIdx < N ) {
        C[rowIdx*N + colIdx] = MUL_INTRIN(A[rowIdx*N + colIdx], B[rowIdx*N + colIdx]);
    }
}

__global__ void matrix_div(CAT_GPU_PREC *A, CAT_GPU_PREC *B, CAT_GPU_PREC *C, int N) {

    int colIdx = blockDim.x*blockIdx.x + threadIdx.x;
    int rowIdx = blockDim.y*blockIdx.y + threadIdx.y;

    if( rowIdx < N && colIdx < N ) {
        C[rowIdx*N + colIdx] = DIV_INTRIN(A[rowIdx*N + colIdx], B[rowIdx*N + colIdx]);
    }
}

__global__ void matrix_sqrt(CAT_GPU_PREC *A, CAT_GPU_PREC *B, CAT_GPU_PREC *C, int N) {

    int colIdx = blockDim.x*blockIdx.x + threadIdx.x;
    int rowIdx = blockDim.y*blockIdx.y + threadIdx.y;

    if( rowIdx < N && colIdx < N ) {
        C[rowIdx*N + colIdx] = SQRT_INTRIN(A[rowIdx*N + colIdx]);
    }
}

__global__ void matrix_fma(CAT_GPU_PREC *A, CAT_GPU_PREC *B, CAT_GPU_PREC *C, int N) {
 
    int colIdx = blockDim.x*blockIdx.x + threadIdx.x;
    int rowIdx = blockDim.y*blockIdx.y + threadIdx.y;
 
    if( rowIdx < N && colIdx < N ) {
        C[rowIdx*N + colIdx] = FMA_INTRIN(A[rowIdx*N + colIdx], B[rowIdx*N + colIdx], C[rowIdx*N + colIdx]);
    }
}

#endif // End of GPU_NVIDIA

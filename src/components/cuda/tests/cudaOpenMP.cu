/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Multi-GPU sample using OpenMP for threading on the CPU side
 * needs a compiler that supports OpenMP 2.0
 */

#ifdef PAPI
#include <papi.h>
#include "papi_test.h"

#define PAPI_CALL(apiFuncCall)                                          \
do {                                                                           \
    int _status = apiFuncCall;                                         \
    if (_status != PAPI_OK) {                                              \
        fprintf(stderr, "error: function %s failed.", #apiFuncCall);  \
        test_fail(__FILE__, __LINE__, "", _status);  \
    }                                                                          \
} while (0)

#endif


#include "gpu_work.h"
#include <omp.h>
#include <stdio.h>  // stdio functions are used since C++ streams aren't necessarily thread safe

#define PRINT(quiet, format, args...) {if (!quiet) {fprintf(stderr, format, ## args);}}
int quiet;

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define MAX_THREADS (32)

/** @class add_events_from_command_line
  * @brief Try and add each event provided on the command line by the user.
  *
  * @param EventSet
  *   A PAPI eventset.
  * @param totalEventCount
  *   Number of events from the command line.
  * @param gpu_id
  *   NVIDIA device index.
  * @param **eventNamesFromCommandLine
  *   Events provided on the command line.
  * @param *numEventsSuccessfullyAdded
  *   Total number of successfully added events.
  * @param **eventsSuccessfullyAdded
  *   Events that we are able to add to the EventSet.
  * @param *numMultipassEvents
  *   Counter to see if a multiple pass event was provided on the command line.
*/
static void add_events_from_command_line(int EventSet, int totalEventCount, int gpu_id, char **eventNamesFromCommandLine, int *numEventsSuccessfullyAdded, char **eventsSuccessfullyAdded, int *numMultipassEvents)
{
    int i;
    for (i = 0; i < totalEventCount; i++) {
        char tmpEventName[PAPI_MAX_STR_LEN];
        int strLen = snprintf(tmpEventName, PAPI_MAX_STR_LEN, "%s:device=%d", eventNamesFromCommandLine[i], gpu_id);
        if (strLen < 0 || strLen >= PAPI_MAX_STR_LEN) {
            fprintf(stderr, "Failed to fully write event name with appended device qualifier.\n");
            test_skip(__FILE__, __LINE__, "", 0);
        }

        int papi_errno = PAPI_add_named_event(EventSet, tmpEventName);
        if (papi_errno != PAPI_OK) {
            if (papi_errno != PAPI_EMULPASS) {
                fprintf(stderr, "Unable to add event %s to the EventSet with error code %d.\n", tmpEventName, papi_errno);
                test_skip(__FILE__, __LINE__, "", 0);
            }

            // Handle multiple pass events
            (*numMultipassEvents)++;
            continue;
        }

        // Handle successfully added events
        strLen = snprintf(eventsSuccessfullyAdded[(*numEventsSuccessfullyAdded)], PAPI_MAX_STR_LEN, "%s", tmpEventName);
        if (strLen < 0 || strLen >= PAPI_MAX_STR_LEN) {
            fprintf(stderr, "Failed to fully write successfully added event.\n");
            test_skip(__FILE__, __LINE__, "", 0);
        }
        (*numEventsSuccessfullyAdded)++;
    }

    return;
}

int main(int argc, char *argv[])
{
    quiet = 0;
#ifdef PAPI
    char *test_quiet = getenv("PAPI_CUDA_TEST_QUIET");
    if (test_quiet)
        quiet = (int) strtol(test_quiet, (char**) NULL, 10);

    int event_count = argc - 1;
    /* if no events passed at command line, just report test skipped. */
    if (event_count == 0) {
        fprintf(stderr, "No eventnames specified at command line.\n");
        test_skip(__FILE__, __LINE__, "", 0);
    }
#endif

    int num_gpus = 0, i;
    hipCtx_t ctx_arr[MAX_THREADS];

    RUNTIME_API_CALL(hipGetDeviceCount(&num_gpus));  // determine the number of CUDA capable GPUs

    if (num_gpus < 1) {
        fprintf(stderr, "no CUDA capable devices were detected\n");
#ifdef PAPI
        test_skip(__FILE__, __LINE__, "", 0);
#endif
        return 0;
    }
    /////////////////////////////////////////////////////////////////
    // display CPU and GPU configuration
    //
    PRINT(quiet, "number of host CPUs:\t%d\n", omp_get_num_procs());
    PRINT(quiet, "number of CUDA devices:\t%d\n", num_gpus);

    for (i = 0; i < num_gpus; i++) {
        hipDeviceProp_t dprop;
        RUNTIME_API_CALL(hipGetDeviceProperties(&dprop, i));
        PRINT(quiet, "   %d: %s\n", i, dprop.name);
    }
    int num_threads = (num_gpus > MAX_THREADS) ? MAX_THREADS : num_gpus;
    // Create a gpu context for every thread
    for (i=0; i < num_threads; i++) {
        int flags = 0;
        hipDevice_t device = i % num_gpus;
#if defined(CUDA_TOOLKIT_GE_13)
        DRIVER_API_CALL( hipCtxCreate(&(ctx_arr[i]), (CUctxCreateParams*)0, flags, device) );
#else
        DRIVER_API_CALL( hipCtxCreate(&(ctx_arr[i]), flags, device) );
#endif
        DRIVER_API_CALL(hipCtxPopCurrent(&(ctx_arr[i])));
    }

    PRINT(quiet, "---------------------------\n");
#ifdef PAPI
    int papi_errno = PAPI_library_init( PAPI_VER_CURRENT );
    if ( papi_errno != PAPI_VER_CURRENT ) {
        test_fail(__FILE__, __LINE__, "PAPI_library_init failed.", 0);
    }
    PAPI_CALL(PAPI_thread_init((unsigned long (*)(void)) omp_get_thread_num));
#endif

    omp_lock_t lock;
    omp_init_lock(&lock);

    PRINT(quiet, "Launching %d threads.\n", num_threads);
    omp_set_num_threads(num_threads);  // create as many CPU threads as there are CUDA devices
    int numMultipassEvents = 0;
#pragma omp parallel
    {
        unsigned int cpu_thread_id = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();
        PRINT(quiet, "cpu_thread_id %u, num_cpu_threads %u, num_threads %d, num_gpus %d\n", cpu_thread_id, num_cpu_threads, num_threads, num_gpus);

        DRIVER_API_CALL(hipCtxPushCurrent(ctx_arr[cpu_thread_id]));
#ifdef PAPI
        int gpu_id = cpu_thread_id % num_gpus;
        int EventSet = PAPI_NULL;
        long long values[MAX_THREADS];
        int j, errno;
        PAPI_CALL(PAPI_create_eventset(&EventSet));
        PRINT(quiet, "CPU thread %d (of %d) uses CUDA device %d with context %p @ eventset %d\n", cpu_thread_id, num_cpu_threads, gpu_id, ctx_arr[cpu_thread_id], EventSet);

        int numEventsSuccessfullyAdded = 0;
        char **eventsSuccessfullyAdded, **metricNames = argv + 1;
        eventsSuccessfullyAdded = (char **) malloc(event_count * sizeof(char *));
        if (eventsSuccessfullyAdded == NULL) {
            fprintf(stderr, "Failed to allocate memory for successfully added events.\n");
            test_skip(__FILE__, __LINE__, "", 0);
        }
        for (i = 0; i < event_count; i++) {
            eventsSuccessfullyAdded[i] = (char *) malloc(PAPI_MAX_STR_LEN * sizeof(char));
            if (eventsSuccessfullyAdded[i] == NULL) {
                fprintf(stderr, "Failed to allocate memory for command line argument.\n");
                test_skip(__FILE__, __LINE__, "", 0);
            }
        }

        add_events_from_command_line(EventSet, event_count, gpu_id, metricNames, &numEventsSuccessfullyAdded, eventsSuccessfullyAdded, &numMultipassEvents);

        // Only multiple pass events were provided on the command line
        if (numEventsSuccessfullyAdded == 0) {
            fprintf(stderr, "Events provided on the command line could not be added to an EventSet as they require multiple passes.\n");
            test_skip(__FILE__, __LINE__, "", 0);
        }

        PAPI_CALL(PAPI_start(EventSet));
#endif
        VectorAddSubtract(50000*(cpu_thread_id+1), quiet);  // gpu work
#ifdef PAPI
        PAPI_CALL(PAPI_stop(EventSet, values));

        PRINT(quiet, "User measured values.\n");
        for (j = 0; j < numEventsSuccessfullyAdded; j++) {
            PRINT(quiet, "%s\t\t%lld\n", eventsSuccessfullyAdded[j], values[j]);
        }

        // Free allocated memory
        for (i = 0; i < event_count; i++) {
            free(eventsSuccessfullyAdded[i]);
        }
        free(eventsSuccessfullyAdded);

        DRIVER_API_CALL(hipCtxPopCurrent(&(ctx_arr[gpu_id])));

        errno = PAPI_cleanup_eventset(EventSet);
        if (errno != PAPI_OK) {
            fprintf(stderr, "PAPI_cleanup_eventset(%d) failed with error %d", EventSet, errno);
            test_fail(__FILE__, __LINE__, "", errno);
        }
        PAPI_CALL(PAPI_destroy_eventset(&EventSet));
#endif
    }  // omp parallel region end

    for (i = 0; i < num_threads; i++) {
        DRIVER_API_CALL(hipCtxDestroy(ctx_arr[i]));
    }

    if (hipSuccess != hipGetLastError())
        fprintf(stderr, "%s\n", hipGetErrorString(hipGetLastError()));

    omp_destroy_lock(&lock);
#ifdef PAPI
    PAPI_shutdown();

    // Output a note that a multiple pass event was provided on the command line
    if (numMultipassEvents > 0) {
        PRINT(quiet, "\033[0;33mNOTE: From the events provided on the command line, an event or events requiring multiple passes was detected and not added to the EventSet. Check your events with utils/papi_native_avail.\n\033[0m");
    }

    test_pass(__FILE__);
#endif
    return 0;
}
